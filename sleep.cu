#include "hip/hip_runtime.h"
#include <stdio.h>

//This file will run nkernel many kernels concurrently and each
//  of them will sleep for kernel_time ms. These two numbers can
//  be passed in as parameters, currently just list the two integers
//  in the command line with nkernels first then kernel_time.

//This file is intended to be used for measuring the overhead in creating
//  kernels and using GPGPUs



// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void clock_block(int kernel_time, int clockRate)
{ 
    int finish_clock;
    int start_time;
    for(int temp=0; temp<kernel_time; temp++){
        start_time = clock();
        finish_clock = start_time + clockRate;
        bool wrapped = finish_clock < start_time;
        while( clock() < finish_clock || wrapped) wrapped = clock()>0 && wrapped;
    }
}

int main(int argc, char **argv)
{
    //Default values
    int nkernels = 16;              // number of concurrent kernels
    int nstreams = nkernels + 1;    // use one more stream than concurrent kernel
    int kernel_time = 2500;         // time the kernel should run in ms
    int cuda_device = 0;

    if( argc>2 ){
        nkernels = atoi(argv[1]);       //could be used to pass in parameters
        kernel_time = atoi(argv[2]);
    }

    //Getting device information, because we need clock_rate later
    hipDeviceProp_t deviceProp;
    hipGetDevice(&cuda_device);	
    hipGetDeviceProperties(&deviceProp, cuda_device);


    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 1; i < nstreams; i++)
        hipStreamCreate(&(streams[i]));

    //////////////////////////////////////////////////////////////////////

    int clockRate = deviceProp.clockRate; 

    //I am starting this at i=1 because the default stream is 0.
    for( int i=1; i<nkernels+1; ++i)
    {
        printf("starting kernel:  %d\n", i);
        clock_block<<<1,1,1,streams[i]>>>(kernel_time, clockRate);
    }

    //Find an errors that the gpu kernels had
    hipError_t cuda_error = hipDeviceSynchronize();

    if(cuda_error==hipSuccess){
        printf( "  Running the concurrentKernels was a success\n");
    }else{
        if(cuda_error==hipErrorLaunchTimeOut ){
            printf( "  A thread was stopped for reaching time limit\n" );
        }else{
            printf( "  An error happened while running the wait\n" );
        }
    }

    // release resources
    for(int i = 1; i < nstreams; i++)
        hipStreamDestroy(streams[i]); 
 
    free(streams);
  return 0;    
}
