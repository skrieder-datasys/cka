#include "hip/hip_runtime.h"
#include <stdio.h>

//This file will run nkernel many kernels concurrently and each
//  of them will sleep for Kernel_time ms. This will work correctly
//  until clock() ~ 2.15 billion clicks. On my system, where the GPU
//  run at 1.56 GHz, this happens in 1.37 seconds.
//Therefore this cannot be used to run tests that will call clock()
//  after more than ~1.37 seconds.


// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void clock_block(int kernel_time, int clockRate)
{ 
    int temp;
    int finish_clock;
    for(temp=0; temp<kernel_time; temp++){
        int start_time = clock();
        finish_clock = start_time + clockRate;
        bool wrapped = finish_clock < start_time;
        while( clock() < finish_clock || wrapped) wrapped = clock()>0 && wrapped;
    }
}

int main(int argc, char **argv)
{
    int nkernels = 4;              // number of concurrent kernels
    int nstreams = nkernels + 1;   // use one more stream than concurrent kernel
    int kernel_time = 2500;        // time the kernel should run in ms
    int cuda_device = 0;

    //nkernels = atoi(argv[1]);       //could be used to pass in parameters
    //kernel_time = atoi(argv[2]);



    hipDeviceProp_t deviceProp;
    hipGetDevice(&cuda_device);	

    hipGetDeviceProperties(&deviceProp, cuda_device);

    // allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int i = 1; i < nstreams; i++)
        hipStreamCreate(&(streams[i]));

    //////////////////////////////////////////////////////////////////////

    int clockRate = deviceProp.clockRate; 

    //I am starting this at i=1 because the default stream is 0.
    for( int i=1; i<nkernels+1; ++i)
    {
        printf("starting kernel:  %d\n", i);
        clock_block<<<1,1,1,streams[i]>>>(kernel_time, clockRate);
    }

    //Find an errors that the gpu kernels had
    hipError_t cuda_error = hipDeviceSynchronize();

    if(cuda_error==hipSuccess){
        printf( "  Running the concurrentKernels was a success\n");
    }else{
        if(cuda_error==hipErrorLaunchTimeOut ){
            printf( "  A thread was stopped for reaching time limit\n" );
        }else{
            printf( "  An error happened while running the wait\n" );
        }
    }

    // release resources
    for(int i = 1; i < nstreams; i++)
        hipStreamDestroy(streams[i]); 
 
    free(streams);
  return 0;    
}
